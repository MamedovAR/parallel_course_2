#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void hello(float* a,char* s)
{
	a[0]+=a[1];
	s[0]='h';
	s[1]='e';
	s[2]='l';
	s[3]='l';
	s[4]='o';
	printf("%s\n",s);
}

int main()
{
	float* a=(float*)malloc(8);
	a[0]=2;
	a[1]=3;
	char* s;
	hipMalloc(&s,5);
	float *b;
	hipMalloc(&b,8);
	hipMemcpy(b,a,8,hipMemcpyHostToDevice);
	hello<<<2,5>>>(b,s);
	hipDeviceSynchronize();
	hipMemcpy(a,b,8,hipMemcpyDeviceToHost);
	hipFree(b);
	hipFree(s);
	printf("\n%f %f\n",a[0],a[1]);
	free(a);
	return 0;
}
