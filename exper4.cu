#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipcub/hipcub.hpp>
#include <nvtx3/nvToolsExt.h>
//#include <cub/device/device_radix_sort.h>

int main()
{
    hipError_t stat;
    hipSetDevice(3);
    float* arr=(float*)malloc(10*sizeof(float));
    float* cuarr;
    stat=hipMalloc((void**)&cuarr,10*sizeof(float));
    if(stat!=hipSuccess)
        printf("err 1: %d\n", stat);

    for(int i=0; i<10; i++)
        arr[i]=(float)i;

    stat=hipMemcpy(cuarr,arr,10*sizeof(float),hipMemcpyHostToDevice);
    if(stat!=hipSuccess)
        printf("err 2: %d\n", stat);

    float* m;
    float ans;
    hipMalloc(&m,sizeof(float));
    float* d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;

    stat=hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, cuarr, m, 10);
    if(stat!=hipSuccess)
        printf("err 3: %d\n", stat);

    // Allocate temporary storage
    stat=hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
    if(stat!=hipSuccess)
        printf("err 4: %d\n", stat);

    // Run max-reduction
    stat=hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, cuarr, m, 10);
    if(stat!=hipSuccess)
        printf("err 5: %d\n", stat);
    hipMemcpy((void*)&ans,(void*)m,sizeof(float),hipMemcpyDeviceToHost);
    printf("%f\n",ans);

    // stat=hipMemcpy((void*)arr,(void*)cuarr,10*sizeof(float),hipMemcpyDeviceToHost);
    // if(stat!=hipSuccess)
    //     printf("err 6: %d\n", stat);

    // stat=hipFree((void*)cuarr);
    // if(stat!=hipSuccess)
    //     printf("err 7: %d\n", stat);

    // stat=hipFree((void*)d_temp_storage);
    // if(stat!=hipSuccess)
    //     printf("err 8: %d\n", stat);

    free(arr);
    return 0;
}