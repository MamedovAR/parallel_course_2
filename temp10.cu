#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <nvToolsExt.h>
#include <hip/hip_runtime.h>
#include "/opt/nvidia/hpc_sdk/Linux_x86_64/22.11/math_libs/11.0/targets/x86_64-linux/include/hipblas.h"
#define IDX2F(i,j,ld) (((j)-1)*(ld))+((i)-1)
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

int main(int argc, char** argv)
{
	float a=0;
	int s=0;
	int n=0;
	hipblasStatus_t status;
	hipblasHandle_t handle;
	if(argv[1][1]=='h')
	{
		printf("Put -h to show this.\n");
		printf("Put -a <NUMBER_OF_ACCURACY*10^6> -s <SIZE^2> -n <NUMBER_OF_ITERATION*10^6>.\n");
	}
	else
	{
		for(int k=1; k<argc; k+=2)
		{
			if(argv[k][1]=='a')
				a=(float)atof(argv[k+1]);
			else if(argv[k][1]=='s')
				s=atoi(argv[k+1]);
			else if(argv[k][1]=='n')
				n=atoi(argv[k+1]);
		}
		float* setka = (float*)calloc(s*s,sizeof(float));
		float* arr = (float*)calloc(s*s,sizeof(float));
		setka[0]=10;
		setka[s-1]=20;
		setka[(s-1)*s]=30;
		setka[s*s-1]=20;
		float l1=(10);
		l1/=s;
		float l2=20;
		l2/=s;
		for(int i=1; i<s-1; i++)
		{
			setka[i]=setka[i-1]+l1;
			setka[i*s]+=setka[(i-1)*s]+l2;
			setka[s-1+i*s]+=setka[s-1+(i-1)*s]+l1;
			setka[s*(s-1)+i]+=setka[s*(s-1)+i]+l1;
		}
		int iter=0;
		float err=1;
//		cublasInit();
		status = hipblasCreate(&handle);
#pragma acc data copyin(setka[0:s*s]) create(arr[0:s*s]) copy(s,iter,err,a,n)
		while(err>a && iter<n)
		{
#pragma acc kernels
			{
			iter++;
			err=0;
			}
#pragma acc update host(err)
#pragma acc update host(iter)
#pragma acc parallel loop gang vector vector_length()
			for(int i=0; i<s*s; i++)
			{
				arr[i]=setka[i];
#pragma acc update device(setka[i])
			}
#pragma acc kernels
{
			for(int i=1; i<s-1; i++)
			{
				for(int j=1; j<s-1; j++)
				{
					setka[i+j*(s-1)]=0.25*(arr[i+1+j*(s-1)]+arr[i-1+j*(s-1)]+arr[i+(j-1)*(s-1)]+arr[i+(j+1)*(s-1)]);
#pragma acc update host(setka[i+j*(s-1)])
				}
			}
}
			int nm=0;
			float alpha=-1;
			status=hipblasSaxpy(handle,s*s,&alpha,setka,1,arr,1);
			status=hipblasIsamax(handle,s*s,arr,1,&nm);
			err=arr[nm];
			#pragma acc parallel
			if(iter%100==0 || iter==1)
				printf("%d %f \n",iter, err);
		}
		hipblasDestroy(handle);
//		cublasShutdown();
		printf("Count iterations: %d", iter);
#pragma acc exit data delete(arr[:s*s]) delete (setka[:s*s])
		free(setka);
	}
	return 0;
}
