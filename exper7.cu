#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <stdlib.h>
#include <hipcub/hipcub.hpp>

int main()
{
    hipError_t stat;
    int  num_items;      // e.g., 7
    int  *d_in;          // e.g., [8, 6, 7, 5, 3, 0, 9]
    int  h_in[] = {8, 6, 7, 5, 3, 0, 9};
    int  *d_max;         // e.g., [-]
    stat = hipMalloc((void**)&d_in,7*sizeof(int));
    if(stat!=hipSuccess)
        printf("err 1: %d",stat);
    stat = hipMalloc((void**)&d_max,sizeof(int));
    if(stat!=hipSuccess)
        printf("err 2: %d",stat);
    stat=hipMemcpy(d_in,h_in,7*sizeof(int),hipMemcpyHostToDevice);
    if(stat!=hipSuccess)
        printf("err 3: %d\n", stat);
    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_max, num_items);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run max-reduction
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_max, num_items);
    // d_out <-- [9]
    std::cout << d_max[0];
}