#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include "/opt/nvidia/hpc_sdk/Linux_x86_64/22.11/math_libs/10.2/include/hipblas.h"
#include <stdio.h>
#include <stdlib.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

class Linear
{
    private:
    float* array;
    float* cuarray;
    int w,h;
    public:
    Linear();
    Linear(int a, int b)
    {
        this->array = (float*)malloc(a*sizeof(float));
        hipMalloc(&this->cuarray,a*b);
        hipMalloc(&this->cuout,b);
        this->h=a;
        for(int i=0; i<a; i++)
            for(int j=0; j<b; j++)
                this->array[IDX2C(i,j,this->h)]=float(rand()%255);
        hipMemcpy(this->cuarray,this->array,a*b,hipMemcpyHostToDevice);
    };
    ~Linear()
    {
        hipMemcpy(this->cuarray,this->array,this->w*this->h,hipMemcpyDeviceToHost);
        hipFree(this->cuarray);
        hipFree(this->cuout);
        free(array);
    };

    float* cuout;

    float* forward(float* arr,hipblasHandle_t handle)
    {
        float* skal;
        hipMalloc(&skal,this->h*this->w);
        for(int i=0; i<this->h*this->w; i++)
            skal[i]=1;
        hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,this->h,this->w,1,skal,arr,this->h,this->cuarray,this->w,skal,this->cuout,this->w);
        hipFree(arr);
        return this->cuout;
    };
};

__device__ float sigma(float a)
{
    return (1/(1-exp(-a)));
}

__global__ void sigmoid(float* arr,int n)
{
//    int i=blockIdx.x*blockDim.x+threadIdx.x;
//    int j=blockIdx.y*blockDim.y+threadIdx.y;
    arr[IDX2C(threadIdx.x,blockIdx.x,n)]=sigma(arr[IDX2C(threadIdx.x,blockIdx.x,n)]);
}

class Net
{
    public:
    Linear fc1,fc2,fc3;
    
    Net()
    {
        printf("init\n");
        this->fc1=Linear(32*32,16*16);
        this->fc2=Linear(16*16,4*4);
        this->fc3=Linear(4*4,1);
    };

    float* forward(float* arr)
    {
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        dim3 a(32,32);
        dim3 b(16,16);
        dim3 c(4,4);
        arr=this->fc1.forward(arr,handle);
        sigmoid<<<32,32>>>(arr,32);
        arr=this->fc1.forward(arr,handle);
        sigmoid<<<16,16>>>(arr,16);
        arr=this->fc1.forward(arr,handle);
        sigmoid<<<4,4>>>(arr,4);
        hipblasDestroy(handle);
        return arr;
    };
};

int main()
{
    float* array = (float*)malloc(32*32*sizeof(float));
    for(int i=0; i<32*32; i++)
        array[i]=rand()%255;
    float* cuarr;
    hipMalloc(&cuarr,32*32);
    hipMemcpy(cuarr,array,32*32,hipMemcpyHostToDevice);
    printf("cuda first\n");
    Net net;
    printf("init net\n");
    cuarr = net.forward(cuarr);
    free(array);
    array=(float*)malloc(sizeof(float));
    hipMemcpy(array,cuarr,1,hipMemcpyDeviceToHost);
    hipFree(cuarr);
    printf("%f\n",array);
    free(array);
    return 0;
}
