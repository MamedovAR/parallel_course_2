#include <ctime>
#include <cmath>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
//Класс nn.Linear
class Linear
{
    private:
    double* array;
    double* cuarray;
    int w,h;
    public:
//Инициализация
    Linear(){
        this->w=0;
        this->h=0;
        this->array=NULL;
        this->cuarray=NULL;
//        this->cuout=NULL;
    };
    
    Linear(const Linear& ln)
    {
        hipMemcpy(this->cuarray,ln.cuarray,ln.w*ln.h,hipMemcpyDeviceToDevice);
        this->array=ln.array;
        this->h=ln.h;
        this->w=ln.w;
    };
    ~Linear()
    {
        hipFree(this->cuarray);
//        cudaFree(this->cuout);
        free(array);
    };

    Linear& operator=(const Linear& ln)
    {
	this->cuarray=ln.cuarray;
        this->array=ln.array;
        this->h=ln.h;
        this->w=ln.w;
        return *this;
    };
//Чтение весов из файла
    void initLinear(int a, int b)
    {
        this->array = (double*)malloc(a*b*sizeof(double));
        this->h=a;
        this->w=b;
        FILE* fl;
        if(a==1024)fl = fopen("fc1.bin","rb");
        else if(a==256)fl = fopen("fc2.bin","rb");
        else fl = fopen("fc3.bin","rb");
        float arrf[a*b];
        fread(arrf,sizeof(float),a*b,fl);
        fclose(fl);
        for(int i=0; i<a*b; i++)
            this->array[i]=(double(arrf[i]));//(double)arrf[i];
        hipMalloc(&this->cuarray,a*b*sizeof(double));
        hipMemcpy(this->cuarray,this->array,a*b*sizeof(double),hipMemcpyHostToDevice);
    };
//Прямой проход (умножение на матрицу)
    void forward(double* arr)
    {
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        double* cuout;
        hipMalloc(&cuout,this->w*sizeof(double));
	    double skal=1;
        hipblasDgemv(handle,HIPBLAS_OP_N,this->w,this->h,&skal,this->cuarray,this->w,arr,1,&skal,cuout,1);
        hipFree(arr);
        hipMalloc(&arr,this->w*sizeof(double));
        hipMemcpy(arr,cuout,this->w*sizeof(double),hipMemcpyDeviceToDevice);
        hipblasDestroy(handle);
        hipFree(cuout);
    };
};
//Сигмоида
__device__ double sigma(double x)
{
    return 1 / (1 + std::exp(-x));
}
//Параллельный вызов сигмоиды
__global__ void sigmoid(double* arr,int n)
{
    if(IDX2C(threadIdx.x,blockIdx.x,n)<n*n)
    	arr[IDX2C(threadIdx.x,blockIdx.x,n)]=sigma(arr[IDX2C(threadIdx.x,blockIdx.x,n)]);
}
//Класс нейронной сети
class Net
{
    public:
    Linear fc1,fc2,fc3;
//Инициализация
    Net()
    {
        this->fc1.initLinear(32*32,16*16);
        this->fc2.initLinear(16*16,4*4);
        this->fc3.initLinear(4*4,1);
    };
//Прямой проход: полносвязный линейный слой и сигмоида в качестве функции активации
    void forward(double* arr)
    {
        double arr1[32*32];
        hipMemcpy(arr1,arr,32*32*sizeof(double),hipMemcpyDeviceToHost);
        //printf("first layer\n");
        this->fc1.forward(arr);
        double arr2[16*16];
        hipMemcpy(arr2,arr,16*16*sizeof(double),hipMemcpyDeviceToHost);
        //for(int i=0; i<16; i++)
            //printf("%f\n",arr2[i]);
        sigmoid<<<16,32>>>(arr,16);
        //printf("second layer\n");
        this->fc2.forward(arr);
        hipMemcpy(arr2,arr,4*4*sizeof(double),hipMemcpyDeviceToHost);
        //for(int i=0; i<16; i++)
            //printf("%f\n",arr2[i]);
        sigmoid<<<4,32>>>(arr,4);
        //printf("third layer\n");
        this->fc3.forward(arr);
        hipMemcpy(arr2,arr,sizeof(double),hipMemcpyDeviceToHost);
        //for(int i=0; i<1; i++)
            //printf("%f\n",arr2[i]);
        arr2[0]=1/(1-exp(-arr2[0]));
//        //printf("%f",arr2[0]);
        sigmoid<<<1,32>>>(arr,1);
    };
};

int main()
{
//Замер времени
    std::time_t result = std::time(nullptr);
    double* array = (double*)malloc(32*32*sizeof(double));
//    float arrf[32*32];
//Чтение входных данных из файла (сгенерировано случайно)
    FILE* fl;
    fl = fopen("start.bin","rb");
    fread(array,sizeof(double),32*32,fl);
    fclose(fl);
//Преобразование типов
//     for(int i=0; i<32*32; i++)
//     {
// //        array[i]=(std::round(array[i]*1e6)/1e6);
//         array[i]=double(arrf[i]);
//     }
    double* cuarr;
    hipMalloc(&cuarr,32*32*sizeof(double));
    hipMemcpy(cuarr,array,32*32,hipMemcpyHostToDevice);
//Инициализация нейронной сети
    Net net;
//Прямой проход нейронной сети
    net.forward(cuarr);
    free(array);
    array=(double*)malloc(sizeof(double));
    hipMemcpy(array,cuarr,sizeof(double),hipMemcpyDeviceToHost);
//Освобождение ресурсов
    hipFree(cuarr);
    printf("Result: %.4f\n",array[0]-0.2765);
    free(array);
    printf("Time: %d\n",std::time(nullptr) - result);
    return 0;
}
