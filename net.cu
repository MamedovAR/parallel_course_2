#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

class Linear
{
    private:
    float* array;
    float* cuarray;
    int w,h;
    public:
    Linear(){
        this->w=0;
        this->h=0;
        this->array=NULL;
        this->cuarray=NULL;
//        this->cuout=NULL;
    };
    
    Linear(const Linear& ln)
    {
        hipMemcpy(this->cuarray,ln.cuarray,ln.w*ln.h,hipMemcpyDeviceToDevice);
//        cudaMemcpy(this->cuout,ln.cuout,ln.w*ln.h,cudaMemcpyDeviceToDevice);
        this->array=ln.array;
        this->h=ln.h;
        this->w=ln.w;
    };
    ~Linear()
    {
        hipFree(this->cuarray);
//        cudaFree(this->cuout);
        free(array);
    };

    Linear& operator=(const Linear& ln)
    {
//	cudaMalloc(&this->cuarray,ln.h*ln.w*sizeof(float));
//        cudaMemcpy(this->cuarray,ln.cuarray,ln.w*ln.h,cudaMemcpyDeviceToDevice);
//        cudaMemcpy(this->cuout,ln.cuout,ln.w*ln.h,cudaMemcpyDeviceToDevice);
	this->cuarray=ln.cuarray;
        this->array=ln.array;
        this->h=ln.h;
        this->w=ln.w;
        return *this;
    };
    void initLinear(int a, int b)
    {
        this->array = (float*)malloc(a*b*sizeof(float));
        this->h=a;
        this->w=b;
	FILE* fl;
	if(a==1024)fl = fopen("fc1.bin","rb");
	else if(a==256)fl = fopen("fc2.bin","rb");
	else fl = fopen("fc3.bin","rb");
	fread(this->array,sizeof(float),a*b,fl);
	fclose(fl);
//	for(int i=0; i<16; i++)
//		printf("%f\n",this->array[i]);
        hipMalloc(&this->cuarray,a*b*sizeof(float));
//        for(int i=0; i<a; i++)
//            for(int j=0; j<b; j++)
//	    {
//                this->array[IDX2C(j,i,a*b)]=float(rand()%255)/255;
//		printf("%f ",this->array[IDX2C(j,i,a*b)]);
//	    }
        hipMemcpy(this->cuarray,this->array,a*b*sizeof(float),hipMemcpyHostToDevice);
    };

    void forward(float* arr)
    {
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        float* cuout;
        hipMalloc(&cuout,this->w*sizeof(float));
//	float dop2[this->w];
//	cudaMemcpy(dop2,this->cuarray,this->w*sizeof(float),cudaMemcpyDeviceToHost);
//	printf("weights\n");
//	for(int i=0; i<4; i++)
//		printf("%f\n",dop2[i]);
	float skal=1;
//        printf("start fc\n");
//	cublasStatus_t stat;
        hipblasSgemv(handle,HIPBLAS_OP_N,this->w,this->h,&skal,this->cuarray,this->w,arr,1,&skal,cuout,1);
//        printf("%d\n",stat);
        hipFree(arr);
        hipMalloc(&arr,this->w*sizeof(float));
        std::swap(arr,cuout);
        hipblasDestroy(handle);
        hipFree(cuout);
    };
};

__device__ float sigma(float a)
{
    return (1/(1-exp(-a)));
}

__global__ void sigmoid(float* arr,int n)
{
//    int i=blockIdx.x*blockDim.x+threadIdx.x;
//    int j=blockIdx.y*blockDim.y+threadIdx.y;
    if(IDX2C(threadIdx.x,blockIdx.x,n)<n*n)
    	arr[IDX2C(threadIdx.x,blockIdx.x,n)]=sigma(arr[IDX2C(threadIdx.x,blockIdx.x,n)]);
}

class Net
{
    public:
    Linear fc1,fc2,fc3;
    
    Net()
    {
//        printf("init\n");
	this->fc1.initLinear(32*32,16*16);
	this->fc2.initLinear(16*16,4*4);
	this->fc3.initLinear(4*4,1);
//        this->fc1=Linear(32*32,16*16);
//        this->fc2=Linear(16*16,4*4);
//        this->fc3=Linear(4*4,1);
    };

    void forward(float* arr)
    {
        float arr1[32*32];
//        cudaMemcpy(arr1,arr,32*32*sizeof(float),cudaMemcpyDeviceToHost);
//	for(int i=0; i<32; i++)
//		std::cout << arr1[i] << "\n";
        this->fc1.forward(arr);
//       	cudaMemcpy(arr1,arr,32*32*sizeof(float),cudaMemcpyDeviceToHost);
//	for(int i=0; i<32; i++)
//		std::cout << arr1[i] << "\n";
//        printf("first fc---------------------------------------------------------\n");
        sigmoid<<<16,16>>>(arr,16);
//	float arr2[16*16];
//	cudaMemcpy(arr2,arr,16*16*sizeof(float),cudaMemcpyDeviceToHost);
//        for(int i=0; i<16; i++)
//            printf("%f\n",arr2[i]);
        this->fc2.forward(arr);
        sigmoid<<<4,4>>>(arr,4);
//	float arr3[4*4];
//	cudaMemcpy(arr3,arr,4*4*sizeof(float),cudaMemcpyDeviceToHost);
//        for(int i=0; i<4; i++)
//            printf("%f\n",arr3[i]);
        this->fc3.forward(arr);
        sigmoid<<<1,1>>>(arr,1);
	float arr0;
//	cudaMemcpy(&arr0,arr,sizeof(float),cudaMemcpyDeviceToHost);
//        printf("%f\n",arr0);
//	printf("end forward net\n");
    };
};

int main()
{
    float* array = (float*)malloc(32*32*sizeof(float));
    FILE* fl;
    fl = fopen("start.bin","rb");
    fread(array,sizeof(float),32*32,fl);
    fclose(fl);
//    for(int i=0; i<32*32; i++)
//    {
//        array[i]=(float(rand()%255))/255;
//	std::cout << array[i] << '\n';
//    }
    float* cuarr;
    hipMalloc(&cuarr,32*32*sizeof(float));
    hipMemcpy(cuarr,array,32*32,hipMemcpyHostToDevice);
//    printf("cuda first\n");
    Net net;
//    printf("init net\n");
    net.forward(cuarr);
    free(array);
    array=(float*)malloc(sizeof(float));
    hipMemcpy(array,cuarr,sizeof(float),hipMemcpyDeviceToHost);
//    std::cout << array[0] << '\n';
    hipFree(cuarr);
    printf("%f\n",array[0]);
    free(array);
    return 0;
}
