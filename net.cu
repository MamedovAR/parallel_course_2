#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include "/opt/nvidia/hpc_sdk/Linux_x86_64/22.11/math_libs/10.2/include/hipblas.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

class Linear
{
    private:
    float* array;
    float* cuarray;
    float* cuout;
    int w,h;
    public:
    Linear(){
        this->w=0;
        this->h=0;
        this->array=NULL;
        this->cuarray=NULL;
//        this->cuout=NULL;
    };
    Linear(int a, int b)
    {
        this->array = (float*)malloc(a*b*sizeof(float));
        this->h=a;
        this->w=b;
        hipMalloc(&this->cuout,this->w*sizeof(float));
        for(int i=0; i<a; i++)
            for(int j=0; j<b; j++)
                this->array[IDX2C(i,j,this->h)]=float(rand()%255);
//        hipMalloc(&cuarray,a*b*sizeof(float));
        hipMemcpy(this->cuarray,this->array,a*b*sizeof(float),hipMemcpyHostToDevice);
    };
    Linear(const Linear& ln)
    {
        hipMemcpy(this->cuarray,ln.cuarray,ln.w*ln.h,hipMemcpyDeviceToDevice);
//        hipMemcpy(this->cuout,ln.cuout,ln.w*ln.h,hipMemcpyDeviceToDevice);
        this->array=ln.array;
        this->h=ln.h;
        this->w=ln.w;
    };
    ~Linear()
    {
        hipFree(this->cuarray);
//        hipFree(this->cuout);
        free(array);
    };

    Linear& operator=(const Linear& ln)
    {
        hipMemcpy(this->cuarray,ln.cuarray,ln.w*ln.h,hipMemcpyDeviceToDevice);
//        hipMemcpy(this->cuout,ln.cuout,ln.w*ln.h,hipMemcpyDeviceToDevice);
        this->array=ln.array;
        this->h=ln.h;
        this->w=ln.w;
        return *this;
    };

    void forward(float* arr)
    {
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        float* skal;
        hipMalloc(&skal,this->h*this->w*sizeof(float));
        float* cuout;
        hipMalloc(&cuout,this->w*sizeof(float));
        float dop[this->w*this->h];
        for(int i=0; i<this->h*this->w; i++)
            dop[i]=1;
        hipMemcpy(skal,dop,this->w*this->h*sizeof(float),hipMemcpyHostToDevice);
        printf("start fc\n");
        hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,this->h,this->w,1,skal,arr,this->h,this->cuarray,this->w,skal,cuout,this->w);
        printf("cublas\n");
        hipFree(arr);
        hipMalloc(&arr,this->w*sizeof(float));
        std::swap(arr,cuout);
        hipblasDestroy(handle);
        hipFree(cuout);
    };
};

__device__ float sigma(float a)
{
    return (1/(1-exp(-a)));
}

__global__ void sigmoid(float* arr,int n)
{
//    int i=blockIdx.x*blockDim.x+threadIdx.x;
//    int j=blockIdx.y*blockDim.y+threadIdx.y;
    arr[IDX2C(threadIdx.x,blockIdx.x,n)]=sigma(arr[IDX2C(threadIdx.x,blockIdx.x,n)]);
}

class Net
{
    public:
    Linear fc1,fc2,fc3;
    
    Net()
    {
        printf("init\n");
        this->fc1=Linear(32*32,16*16);
        this->fc2=Linear(16*16,4*4);
        this->fc3=Linear(4*4,1);
    };

    void forward(float* arr)
    {
        float arr1[32*32];
        hipMemcpy(arr1,arr,32*32,hipMemcpyDeviceToHost);
        for(int i=0; i<32; i++)
            printf("%f\n",arr1[i]);
        this->fc1.forward(arr);
        printf("first fc\n");
        sigmoid<<<16,16>>>(arr,16);
        for(int i=0; i<16; i++)
            printf("%f\n",arr[i]);
        this->fc1.forward(arr);
        sigmoid<<<4,4>>>(arr,4);
        for(int i=0; i<4; i++)
            printf("%f\n",arr[i]);
        this->fc1.forward(arr);
        sigmoid<<<1,1>>>(arr,1);
        printf("%f\n",arr[0]);
    };
};

int main()
{
    float* array = (float*)malloc(32*32*sizeof(float));
    for(int i=0; i<32*32; i++)
        array[i]=float(rand()%255)/255;
    float* cuarr;
    hipMalloc(&cuarr,32*32);
    hipMemcpy(cuarr,array,32*32,hipMemcpyHostToDevice);
    printf("cuda first\n");
    Net net;
    printf("init net\n");
    net.forward(cuarr);
    free(array);
    array=(float*)malloc(sizeof(float));
    hipMemcpy(array,cuarr,1,hipMemcpyDeviceToHost);
//    std::cout << array[0];
    hipFree(cuarr);
    printf("%f\n",array[0]);
    free(array);
    return 0;
}
