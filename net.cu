#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include "/opt/nvidia/hpc_sdk/Linux_x86_64/22.11/math_libs/10.2/include/hipblas.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

class Linear
{
    private:
    double* array;
    double* cuarray;
    double* cuout;
    int w,h;
    public:
    Linear(){
        this->w=0;
        this->h=0;
        this->array=NULL;
        this->cuarray=NULL;
//        this->cuout=NULL;
    };
    Linear(int a, int b)
    {
        this->array = (double*)malloc(a*b*sizeof(double));
        this->h=a;
        this->w=b;
        hipMalloc(&this->cuout,this->w*sizeof(double));
        for(int i=0; i<a; i++)
            for(int j=0; j<b; j++)
                this->array[IDX2C(i,j,this->h)]=double(rand()%255)/255;
//        hipMalloc(&cuarray,a*b*sizeof(double));
        hipMemcpy(this->cuarray,this->array,a*b*sizeof(double),hipMemcpyHostToDevice);
    };
    Linear(const Linear& ln)
    {
        hipMemcpy(this->cuarray,ln.cuarray,ln.w*ln.h,hipMemcpyDeviceToDevice);
//        hipMemcpy(this->cuout,ln.cuout,ln.w*ln.h,hipMemcpyDeviceToDevice);
        this->array=ln.array;
        this->h=ln.h;
        this->w=ln.w;
    };
    ~Linear()
    {
        hipFree(this->cuarray);
//        hipFree(this->cuout);
        free(array);
    };

    Linear& operator=(const Linear& ln)
    {
        hipMemcpy(this->cuarray,ln.cuarray,ln.w*ln.h,hipMemcpyDeviceToDevice);
//        hipMemcpy(this->cuout,ln.cuout,ln.w*ln.h,hipMemcpyDeviceToDevice);
        this->array=ln.array;
        this->h=ln.h;
        this->w=ln.w;
        return *this;
    };

    void forward(double* arr)
    {
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        double* cuout;
        hipMalloc(&cuout,this->w*sizeof(double));
//		double dop2[this->w];
//		hipMemcpy(dop2,cuout,this->w*sizeof(double),hipMemcpyDeviceToHost);
//		for(int i=0; i<this->h; i++)
//			printf("%d\n",this->cuarray[i]);
		double skal=1;
        printf("start fc\n");
        hipblasDgemv(handle,HIPBLAS_OP_T,this->w,this->h,&skal,arr,1,this->cuarray,1,&skal,cuout,1);
        printf("cublas\n");
        hipFree(arr);
        hipMalloc(&arr,this->w*sizeof(double));
        std::swap(arr,cuout);
        hipblasDestroy(handle);
        hipFree(cuout);
    };
};

__device__ double sigma(double a)
{
    return (1/(1-exp(-a)));
}

__global__ void sigmoid(double* arr,int n)
{
//    int i=blockIdx.x*blockDim.x+threadIdx.x;
//    int j=blockIdx.y*blockDim.y+threadIdx.y;
    arr[IDX2C(threadIdx.x,blockIdx.x,n)]=sigma(arr[IDX2C(threadIdx.x,blockIdx.x,n)]);
}

class Net
{
    public:
    Linear fc1,fc2,fc3;
    
    Net()
    {
        printf("init\n");
        this->fc1=Linear(32*32,16*16);
        this->fc2=Linear(16*16,4*4);
        this->fc3=Linear(4*4,1);
    };

    void forward(double* arr)
    {
        double arr1[32*32];
        hipMemcpy(arr1,arr,32*32*sizeof(double),hipMemcpyDeviceToHost);
        for(int i=0; i<32; i++)
            printf("%f\n",arr1[i]);
        this->fc1.forward(arr);
        printf("first fc\n");
        sigmoid<<<16,16>>>(arr,16);
		double arr2[16*16];
		hipMemcpy(arr2,arr,16*16*sizeof(double),hipMemcpyDeviceToHost);
        for(int i=0; i<16; i++)
            printf("%f\n",arr2[i]);
        this->fc1.forward(arr);
        sigmoid<<<4,4>>>(arr,4);
		double arr3[4*4];
		hipMemcpy(arr3,arr,4*4*sizeof(double),hipMemcpyDeviceToHost);
        for(int i=0; i<4; i++)
            printf("%f\n",arr3[i]);
        this->fc1.forward(arr);
        sigmoid<<<1,1>>>(arr,1);
		double arr0;
		hipMemcpy(&arr0,arr,sizeof(double),hipMemcpyDeviceToHost);
        printf("%f\n",arr0);
		printf("end forward net\n");
    };
};

int main()
{
    double* array = (double*)malloc(32*32*sizeof(double));
    for(int i=0; i<32*32; i++)
	{
        array[i]=(double(rand()%255))/255;
		std::cout << array[i] << '\n';
	}
    double* cuarr;
    hipMalloc(&cuarr,32*32*sizeof(double));
    hipMemcpy(cuarr,array,32*32,hipMemcpyHostToDevice);
    printf("cuda first\n");
    Net net;
    printf("init net\n");
    net.forward(cuarr);
    free(array);
    array=(double*)malloc(sizeof(double));
    hipMemcpy(array,cuarr,sizeof(double),hipMemcpyDeviceToHost);
//    std::cout << array[0] << '\n';
    hipFree(cuarr);
    printf("%f\n",array);
//    free(array);
    return 0;
}
