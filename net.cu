#include <ctime>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
//Класс nn.Linear
class Linear
{
    private:
    double* array;
    double* cuarray;
    int w,h;
    public:
    Linear(){
        this->w=0;
        this->h=0;
        this->array=NULL;
        this->cuarray=NULL;
//        this->cuout=NULL;
    };
    
    Linear(const Linear& ln)
    {
        hipMemcpy(this->cuarray,ln.cuarray,ln.w*ln.h,hipMemcpyDeviceToDevice);
        this->array=ln.array;
        this->h=ln.h;
        this->w=ln.w;
    };
    ~Linear()
    {
        hipFree(this->cuarray);
//        cudaFree(this->cuout);
        free(array);
    };

    Linear& operator=(const Linear& ln)
    {
	this->cuarray=ln.cuarray;
        this->array=ln.array;
        this->h=ln.h;
        this->w=ln.w;
        return *this;
    };
//Чтение весов из файла
    void initLinear(int a, int b)
    {
        this->array = (double*)malloc(a*b*sizeof(double));
        this->h=a;
        this->w=b;
        FILE* fl;
        if(a==1024)fl = fopen("fc1.bin","rb");
        else if(a==256)fl = fopen("fc2.bin","rb");
        else fl = fopen("fc3.bin","rb");
        float arrf[a*b];
        fread(arrf,sizeof(float),a*b,fl);
        fclose(fl);
        for(int i=0; i<a*b; i++)
            this->array[i]=(double)arrf[i];
        hipMalloc(&this->cuarray,a*b*sizeof(double));
        hipMemcpy(this->cuarray,this->array,a*b*sizeof(double),hipMemcpyHostToDevice);
    };

    void forward(double* arr)
    {
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        double* cuout;
        hipMalloc(&cuout,this->w*sizeof(double));
	    double skal=1;
        hipblasDgemv(handle,HIPBLAS_OP_N,this->w,this->h,&skal,this->cuarray,this->w,arr,1,&skal,cuout,1);
        hipFree(arr);
        hipMalloc(&arr,this->w*sizeof(double));
        hipMemcpy(arr,cuout,this->w*sizeof(double),hipMemcpyDeviceToDevice);
        hipblasDestroy(handle);
        hipFree(cuout);
    };
};
//Сигмоида
__device__ double sigma(double a)
{
    return (1/(1-exp(-a)));
}

__global__ void sigmoid(double* arr,int n)
{
    if(IDX2C(threadIdx.x,blockIdx.x,n)<n*n)
    	arr[IDX2C(threadIdx.x,blockIdx.x,n)]=sigma(arr[IDX2C(threadIdx.x,blockIdx.x,n)]);
}
//Класс нейронной сети
class Net
{
    public:
    Linear fc1,fc2,fc3;
    
    Net()
    {
        this->fc1.initLinear(32*32,16*16);
        this->fc2.initLinear(16*16,4*4);
        this->fc3.initLinear(4*4,1);
    };

    void forward(double* arr)
    {
        double arr1[32*32];
        hipMemcpy(arr1,arr,32*32*sizeof(double),hipMemcpyDeviceToHost);
        this->fc1.forward(arr);
        double arr2[16*16];
        hipMemcpy(arr2,arr,16*16*sizeof(double),hipMemcpyDeviceToHost);
        sigmoid<<<16,16>>>(arr,16);
        this->fc2.forward(arr);
        sigmoid<<<4,4>>>(arr,4);
        this->fc3.forward(arr);
        sigmoid<<<1,1>>>(arr,1);
    };
};

int main()
{
    std::time_t result = std::time(nullptr);
    double* array = (double*)malloc(32*32*sizeof(double));
    float arrf[32*32];
    FILE* fl;
    fl = fopen("start.bin","rb");
    fread(arrf,sizeof(double),32*32,fl);
    fclose(fl);
    for(int i=0; i<32*32; i++)
    {
        array[i]=(double(arrf[i]));
    }
    double* cuarr;
    hipMalloc(&cuarr,32*32*sizeof(double));
    hipMemcpy(cuarr,array,32*32,hipMemcpyHostToDevice);
    Net net;
    net.forward(cuarr);
    free(array);
    array=(double*)malloc(sizeof(double));
    hipMemcpy(array,cuarr,sizeof(double),hipMemcpyDeviceToHost);
    hipFree(cuarr);
    printf("Result: %f\n",array[0]);
    free(array);
    printf("Time: %d\n",std::time(nullptr) - result);
    return 0;
}
