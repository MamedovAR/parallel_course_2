#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipcub/hipcub.hpp>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_reduce.cuh>
#define IDX2F(i,j,ld) (((j)-1)*(ld))+((i)-1)
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

__global__ void change(float* setka, float* arr, int s)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i > s && i%s != 0 && i < s*(s - 1)-1 && i%s != s - 1)
		setka[i] = 0.25 * (arr[i-1] + arr[i+1] + arr[i+s] + arr[i-s]);
//	setka[IDX2C(i+threadIdx.x,j+threadIdx.y,s)]=0.25*(arr[IDX2C(i+threadIdx.x,j-1+threadIdx.y,s)]+arr[IDX2C(i+threadIdx.x,j+1+threadIdx.y,s)]+arr[IDX2C(i-1+threadIdx.x,j+threadIdx.y,s)]+arr[IDX2C(i+1+threadIdx.x,j+threadIdx.y,s)]);
}

__global__ void subtract_modulo_kernel(float* d_in1, float* d_in2, float* d_out, int size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < size*size) {
        float diff = d_in1[idx] - d_in2[idx];
        if(diff<0)
          d_out[idx]=-diff;
        else
          d_out[idx]=diff;
    }
}

int main(int argc, char** argv)
{
  float a=0;
  int s=0;
  int n=0;
  if(argv[1][1]=='h')
  {
    printf("Put -h to show this.\n");
    printf("Put -a <NUMBER_OF_ACCURACY*10^6> -s <SIZE^2> -n <NUMBER_OF_ITERATION*10^6>.\n");
  }
  else
  {
    for(int k=1; k<argc; k+=2)
    {
      if(argv[k][1]=='a')
        a=(float)atof(argv[k+1]);
      else if(argv[k][1]=='s')
        s=atoi(argv[k+1]);
      else if(argv[k][1]=='n')
        n=atoi(argv[k+1]);
    }

    float* setka = (float*)calloc(s*s,sizeof(float));
    float* arr = (float*)calloc(s*s,sizeof(float));
    float* arr2 = (float*)calloc(s*s,sizeof(float));

    setka[0]=10;
    setka[s-1]=20;
    setka[(s-1)*s]=20;
    setka[s*s-1]=30;
    arr[0]=10;
    arr[s-1]=20;
    arr[(s-1)*s]=20;
    arr[s*s-1]=30;
    arr2[0]=10;
    arr2[s-1]=20;
    arr2[(s-1)*s]=20;
    arr2[s*s-1]=30;
    float l1=(10);
    l1/=s;
    float l2=20;
    l2/=s;
    int iter=0;
    float err=1;
    for(int i=1; i<s-1; i++)
    {
      setka[i]=setka[i-1]+l1;
      setka[i*s]+=setka[(i-1)*s]+l2;
      setka[s-1+i*s]+=setka[s-1+(i-1)*s]+l1;
      setka[s*(s-1)+i]+=setka[s*(s-1)+i-1]+l1;
      arr[i]=setka[i];
      arr[i*s]=setka[i*s];
      arr[s-1+i*s]=setka[s-1+i*s];
      arr[s*(s-1)+i]=setka[s*(s-1)+i];
    }

    if(s<16)
    {
      for(int i=0; i<s; i++)
      {
        for(int j=0; j<s; j++)
          printf("%f ",setka[i+s*j]);
        printf("\n");
      
      }
    }
  //  hipSetDevice(3);
    float *cusetka;
    float *cuarr;
    float* cuarr2;
    hipError_t stat;
    hipStream_t stream;
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    float* max_value;
    stat=hipMalloc((void**)&cusetka, s*s*sizeof(float));
    if(stat!=hipSuccess)printf("err 1: %d", stat);
    stat=hipMalloc((void**)&cuarr2, s*s*sizeof(float));
    if(stat!=hipSuccess)printf("err 2: %d", stat);
    stat=hipMalloc((void**)&cuarr, s*s*sizeof(float));
    if(stat!=hipSuccess)printf("err 2: %d", stat);
    stat=hipMemcpy(cuarr2, arr2, s*s*sizeof(float), hipMemcpyHostToDevice);
    if(stat!=hipSuccess)printf("err 3: %d", stat);
    stat=hipMemcpy(cusetka, setka, s*s*sizeof(float), hipMemcpyHostToDevice);
    if(stat!=hipSuccess)printf("err 4: %d", stat);
    stat=hipMemcpy(cuarr, arr, s*s*sizeof(float), hipMemcpyHostToDevice);
    if(stat!=hipSuccess)printf("err 5: %d", stat);
    stat=hipMalloc((void**)&max_value, sizeof(float));
    if(stat!=hipSuccess)printf("err 6: %d", stat);
    stat=hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, cuarr2, max_value, s*s);
    if(stat!=hipSuccess)printf("err 7: %d", stat);
    stat=hipMalloc(&d_temp_storage,temp_storage_bytes);
    if(stat!=hipSuccess)printf("err 8: %d", stat);
    float* max_value_h=(float*)malloc(sizeof(float));
    while(err>a && iter<n)
    {
      iter++;
      if(iter%100==1)
        err=0;
        //Этого должно хватить для вычисления массива.
      hipGraph_t graph;
      hipGraphExec_t instance;
//      hipStreamBeginCapture(stream,hipStreamCaptureModeGlobal);
      change<<<s, s, 0 >>>(cusetka, cuarr, s);
//      hipStreamEndCapture(stream, &graph);
//      hipGraphInstantiate(&instance,graph,NULL,NULL,0);
//      change<<<blocksPerGrid, threadsPerBlock >>>(cuarr, cusetka, n);
      if(iter%100==1)
      {
        subtract_modulo_kernel<<<s, s, 0>>>(cusetka, cuarr, cuarr2, s);
        // hipMemcpy(setka,cuarr2,s*s*sizeof(float),hipMemcpyDeviceToHost);
        // if(s<16)
        // {
        //   for(int i=0; i<s; i++)
        //   {
        //     for(int j=0; j<s; j++)
        //       printf("%f ",setka[i+s*j]);
        //     printf("\n");
          
        //   }
        // }
        // const int block_size = 256;
        // const int num_blocks = (n + block_size - 1) / block_size;

//        hipDeviceSynchronize();

        stat=hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, cuarr2, max_value, s*s);
        if(stat!=hipSuccess)printf("%d\n",stat);
        hipMemcpy(max_value_h,max_value,sizeof(float),hipMemcpyDeviceToHost);
        err=max_value_h[0];
        printf("%d %f\n", iter, err);
      }

      float* dop;
      dop = cuarr;
      cuarr=cusetka;
      cusetka = dop;
      // hipMemcpy(cuarr2,cuarr,s*s*sizeof(float),hipMemcpyDeviceToDevice);
      // hipMemcpy(cuarr,cusetka,s*s*sizeof(float),hipMemcpyDeviceToDevice);
      // hipMemcpy(cusetka,cuarr2,s*s*sizeof(float),hipMemcpyDeviceToDevice);
      //std::swap(cuarr,cusetka);
    }
    hipMemcpy(setka,cusetka,s*s*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(arr, cuarr, s*s*sizeof(float), hipMemcpyDeviceToHost);
    free(max_value_h);
    hipFree(d_temp_storage);
    hipFree(cusetka);
    hipFree(cuarr);
    printf("Count iterations: %d\nError: %.10f\n", iter,err);
    if(s<16)
    {
      for(int i=0; i<s; i++)
      {
        for(int j=0; j<s; j++)
          printf("%f ",setka[i+s*j]);
        printf("\n");
      }
    }
    free(setka);
    free(arr);
  }
  return 0;
}
