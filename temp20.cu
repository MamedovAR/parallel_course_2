#include "hip/hip_runtime.h"
#include <ctime>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipcub/hipcub.hpp>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_reduce.cuh>
#define IDX2F(i,j,ld) (((j)-1)*(ld))+((i)-1)
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
//Функция для вычисления теплопроводности по пятиточечному шаблону
__global__ void change(double* setka, double* arr, int s)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i > s && i%s != 0 && i < s*(s - 1)-1 && i%s != s - 1)
		setka[i] = 0.25 * (arr[i-1] + arr[i+1] + arr[i+s] + arr[i-s]);
}
//Функция для вычисления разницы между итерациями
__global__ void subtract_modulo_kernel(double* d_in1, double* d_in2, double* d_out, int size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx <= size*size) {
        double diff = d_in1[idx] - d_in2[idx];
        if(diff<0)
          d_out[idx]=-diff;
        else
          d_out[idx]=diff;
    }
}

int main(int argc, char** argv)
{
  double a=0;
  int s=0;
  int n=0;
  if(argv[1][1]=='h')
  {
    printf("Put -h to show this.\n");
    printf("Put -a <NUMBER_OF_ACCURACY*10^6> -s <SIZE^2> -n <NUMBER_OF_ITERATION*10^6>.\n");
  }
  else
  {
    //Парсинг командной строки
    for(int k=1; k<argc; k+=2)
    {
      if(argv[k][1]=='a')
        a=(double)atof(argv[k+1]);
      else if(argv[k][1]=='s')
        s=atoi(argv[k+1]);
      else if(argv[k][1]=='n')
        n=atoi(argv[k+1]);
    }
//Инициализация
    double* setka = (double*)calloc(s*s,sizeof(double));
    double* arr = (double*)calloc(s*s,sizeof(double));
    double* arr2 = (double*)calloc(s*s,sizeof(double));

    setka[0]=10;
    setka[s-1]=20;
    setka[(s-1)*s]=20;
    setka[s*s-1]=30;
    arr[0]=10;
    arr[s-1]=20;
    arr[(s-1)*s]=20;
    arr[s*s-1]=30;
    arr2[0]=10;
    arr2[s-1]=20;
    arr2[(s-1)*s]=20;
    arr2[s*s-1]=30;
    double l1=(10);
    l1/=s-1;
    double l2=20;
    l2/=s-1;
    int iter=0;
    double err=1;
    for(int i=1; i<s-1; i++)
    {
      setka[i]=setka[i-1]+l1;
      setka[i*s]+=setka[(i-1)*s]+l1;
      setka[s-1+i*s]+=setka[s-1+(i-1)*s]+l1;
      setka[s*(s-1)+i]+=setka[s*(s-1)+i-1]+l1;
      arr[i]=setka[i];
      arr[i*s]=setka[i*s];
      arr[s-1+i*s]=setka[s-1+i*s];
      arr[s*(s-1)+i]=setka[s*(s-1)+i];
    }
//Визуализация сеток, меньших, чем 16
    if(s<16)
    {
      for(int i=0; i<s; i++)
      {
        for(int j=0; j<s; j++)
          printf("%f ",setka[i+s*j]);
        printf("\n");
      
      }
    }
  //  hipSetDevice(3);
    double *cusetka;
    double *cuarr;
    double* cuarr2;
    hipError_t stat;
    hipStream_t stream;
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    double* max_value;
    //Выделение памяти на видеокарте
    stat=hipMalloc((void**)&cusetka, s*s*sizeof(double));
    if(stat!=hipSuccess)printf("err 1: %d", stat);
    stat=hipMalloc((void**)&cuarr2, s*s*sizeof(double));
    if(stat!=hipSuccess)printf("err 2: %d", stat);
    stat=hipMalloc((void**)&cuarr, s*s*sizeof(double));
    if(stat!=hipSuccess)printf("err 2: %d", stat);
    stat=hipMemcpy(cuarr2, arr2, s*s*sizeof(double), hipMemcpyHostToDevice);
    if(stat!=hipSuccess)printf("err 3: %d", stat);
    stat=hipMemcpy(cusetka, setka, s*s*sizeof(double), hipMemcpyHostToDevice);
    if(stat!=hipSuccess)printf("err 4: %d", stat);
    stat=hipMemcpy(cuarr, arr, s*s*sizeof(double), hipMemcpyHostToDevice);
    if(stat!=hipSuccess)printf("err 5: %d", stat);
    stat=hipMalloc((void**)&max_value, sizeof(double));
    if(stat!=hipSuccess)printf("err 6: %d", stat);
    //Инициализация hipcub::DeviceReduce::Max
    stat=hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, cuarr2, max_value, s*s);
    if(stat!=hipSuccess)printf("err 7: %d", stat);
    stat=hipMalloc(&d_temp_storage,temp_storage_bytes);
    if(stat!=hipSuccess)printf("err 8: %d", stat);
    double* max_value_h=(double*)malloc(sizeof(double));
    hipGraph_t graph;
//    hipGraphExec_t instance;
    //Основной цикл
	std::time_t result = std::time(nullptr);
    while(err>a && iter<n)
    {
      iter++;
      if(iter%100==1)
        err=0;
        //Этого должно хватить для вычисления массива.
//Вычисление слоя
//Количество потоеков в рамках потоковогоо блока должно быть не больше 1024 и кратно 32.
//Найти количество блоков в сетке , исходя из количества потоков в сетке; исправить заполнение границ; добавить cudaGraph; замерить время внутри кода (библиотеки time).
//Разобраться, почему выводится ноль в результате вычислений. Заменить double на double
      change<<<s, s, 0>>>(cusetka, cuarr, s);
      if(iter%100==1)
      {
        //Вычисление слоя с ошибкой
        subtract_modulo_kernel<<<s, s, 0>>>(cusetka, cuarr, cuarr2, s);
//Вычисление ошибки
        stat=hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, cuarr2, max_value, s*s);
        if(stat!=hipSuccess)printf("%d\n",stat);
        hipMemcpy(max_value_h,max_value,sizeof(double),hipMemcpyDeviceToHost);
        err=max_value_h[0];
        printf("%d %.6f\n", iter, err);
      }
//Копирование
      double* dop;
      dop = cuarr;
      cuarr=cusetka;
      cusetka = dop;
    }
	result = std::time(nullptr) - result;
    //Возвращение данныз на хост
    hipMemcpy(setka,cusetka,s*s*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(arr, cuarr, s*s*sizeof(double), hipMemcpyDeviceToHost);
    free(max_value_h);
    hipFree(d_temp_storage);
    hipFree(cusetka);
    hipFree(cuarr);
    printf("Count iterations: %d\nError: %.8f\nTime: %d\n", iter,err,result);
    if(s<16)
    {
      for(int i=0; i<s; i++)
      {
        for(int j=0; j<s; j++)
          printf("%f ",setka[i+s*j]);
        printf("\n");
      }
    }
    free(setka);
    free(arr);
  }
  return 0;
}
