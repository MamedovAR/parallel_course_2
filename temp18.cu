#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <openacc.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <nvToolsExt.h>
#include <hip/hip_runtime.h>
#include "/opt/nvidia/hpc_sdk/Linux_x86_64/22.11/math_libs/11.0/targets/x86_64-linux/include/hipblas.h"

#define IDX2F(i,j,ld) (((j)-1)*(ld))+((i)-1)
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

int main(int argc, char** argv)
{
  float a=0;
  int s=0;
  int n=0;
  if(argv[1][1]=='h')
  {
    printf("Put -h to show this.\n");
    printf("Put -a <NUMBER_OF_ACCURACY*10^6> -s <SIZE^2> -n <NUMBER_OF_ITERATION*10^6>.\n");
  }
  else
  {
    for(int k=1; k<argc; k+=2)
    {
      if(argv[k][1]=='a')
        a=(float)atof(argv[k+1]);
      else if(argv[k][1]=='s')
        s=atoi(argv[k+1]);
      else if(argv[k][1]=='n')
        n=atoi(argv[k+1]);
    }

    float* setka = (float*)calloc(s*s,sizeof(float));
    float* arr = (float*)calloc(s*s,sizeof(float));
	float* arr2 = (float*)calloc(s*s,sizeof(float));
    float* setka2 = (float*)calloc(s*s,sizeof(float));

    setka[0]=10;
    setka[s-1]=20;
    setka[(s-1)*s]=20;
    setka[s*s-1]=30;
	  arr[0]=10;
    arr[s-1]=20;
    arr[(s-1)*s]=20;
    arr[s*s-1]=30;
	  arr2[0]=10;
    arr2[s-1]=20;
    arr2[(s-1)*s]=20;
    arr2[s*s-1]=30;
    float l1=(10);
    l1/=s;
    float l2=20;
    l2/=s;
    int iter=0;
    float err=1;
    for(int i=1; i<s-1; i++)
    {
      setka[i]=setka[i-1]+l1;
      setka[i*s]+=setka[(i-1)*s]+l2;
      setka[s-1+i*s]+=setka[s-1+(i-1)*s]+l1;
      setka[s*(s-1)+i]+=setka[s*(s-1)+i-1]+l1;
      arr[i]=setka[i];
      arr[i*s]=setka[i*s];
      arr[s-1+i*s]=setka[s-1+i*s];
      arr[s*(s-1)+i]=setka[s*(s-1)+i];
      arr2[i]=setka[i];
      arr2[i*s]=setka[i*s];
      arr2[s-1+i*s]=setka[s-1+i*s];
      arr2[s*(s-1)+i]=setka[s*(s-1)+i];
    }

    if(s<16)
    {
      for(int i=0; i<s; i++)
      {
        for(int j=0; j<s; j++)
          printf("%f ",setka[i+s*j]);
        printf("\n");
      
      }
    }

    hipblasStatus_t status;
    hipblasHandle_t handle;

    #pragma acc data copyin(setka[0:s*s],arr[0:s*s],arr2[0:s*s],setka2[0:s*s],err,iter)
	{
    status = hipblasCreate(&handle);
	  if (status != HIPBLAS_STATUS_SUCCESS)
	  	printf("ERROR: %d!\n",status);
    while(err>a && iter<n)
    {
      iter++;

      #pragma acc kernels
      {
        err=0;
      }
#pragma acc data present(arr, setka)
#pragma acc parallel loop gang num_workers(4) vector_length(128) async(1)
      for(int i=1; i<s-1; i++)
      {
        #pragma acc loop vector 
        for(int j=1; j<s-1; j++)
        {
          arr[IDX2C(i,j,s)]=0.25*(setka[IDX2C(i,j-1,s)]+setka[IDX2C(i,j+1,s)]+setka[IDX2C(i-1,j,s)]+setka[IDX2C(i+1,j,s)]);
        }
      }
      float* dop;
      dop = arr;
      arr=setka;
      setka = dop;

//	  #pragma acc update device(setka,arr) async(1)
      if(iter%100==0 || iter==1)
      {
        int nm=0;
        float alpha[1];
        alpha[0]=-1;
        #pragma acc host_data use_device(arr,setka,arr2)
        {
    //	  #pragma acc update host(setka,arr) async(1)
  //      status=hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
          status=hipblasScopy(handle,s*s,arr,1,arr2,1);
          if (status != HIPBLAS_STATUS_SUCCESS)
            printf("ERROR: %d!\n",status);
          status=hipblasSaxpy(handle,s*s,alpha,setka,1,arr2,1);
          if (status != HIPBLAS_STATUS_SUCCESS)
            printf("ERROR: %d!\n",status);
          status=hipblasIsamax(handle,s*s,arr2,1,&nm);
          if (status != HIPBLAS_STATUS_SUCCESS)
            printf("ERROR: %d!\n",status);
          #pragma acc kernels
          {
          err = fabs(arr[nm-1]);
          }
          #pragma acc update host(err) async(1)
          #pragma acc wait(1)
          printf("%d %f\n",iter, err);
        }
        
//        #pragma acc wait(1) 
      }
    }
    status=hipblasDestroy(handle);
    }
    printf("Count iterations: %d\nError: %.10f\n", iter,err);
	  if (status != HIPBLAS_STATUS_SUCCESS)
	  	printf("ERROR: %d!\n",status);
    if(s<16)
      for(int i=0; i<s; i++)
      {
        for(int j=0; j<s; j++)
          printf("%f ",setka[i+s*j]);
        printf("\n");
      }
    free(arr);
    free(setka);
    free(setka2);
    free(arr2);
  }
  return 0;
}#include <stdio.h>
#include <math.h>
#include <openacc.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <nvToolsExt.h>
#include <hip/hip_runtime.h>
#include "/opt/nvidia/hpc_sdk/Linux_x86_64/22.11/math_libs/11.0/targets/x86_64-linux/include/hipblas.h"

#define IDX2F(i,j,ld) (((j)-1)*(ld))+((i)-1)
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

int main(int argc, char** argv)
{
  float a=0;
  int s=0;
  int n=0;
  if(argv[1][1]=='h')
  {
    printf("Put -h to show this.\n");
    printf("Put -a <NUMBER_OF_ACCURACY*10^6> -s <SIZE^2> -n <NUMBER_OF_ITERATION*10^6>.\n");
  }
  else
  {
    for(int k=1; k<argc; k+=2)
    {
      if(argv[k][1]=='a')
        a=(float)atof(argv[k+1]);
      else if(argv[k][1]=='s')
        s=atoi(argv[k+1]);
      else if(argv[k][1]=='n')
        n=atoi(argv[k+1]);
    }

    float* setka = (float*)calloc(s*s+1,sizeof(float));
    float* arr = (float*)calloc(s*s+1,sizeof(float));
	float* arr2 = (float*)calloc(s*s+1,sizeof(float));
    float* setka2 = (float*)calloc(s*s+1,sizeof(float));
	setka[0]=10;
	setka[s-1]=20;
	setka[(s-1)*s]=20;
	setka[s*s-1]=30;
	arr[0]=10;
	arr[s-1]=20;
	arr[(s-1)*s]=20;
	arr[s*s-1]=30;
	arr2[0]=10;
	arr2[s-1]=20;
	arr2[(s-1)*s]=20;
	arr2[s*s-1]=30;
    float l1=(10);
    l1/=s;
    float l2=20;
    l2/=s;
    int iter=0;
    float err=1;
    for(int i=1; i<s-1; i++)
    {
	    setka[i]=setka[i-1]+l1;
	setka[i*s]+=setka[(i-1)*s]+l2;
	setka[s-1+i*s]+=setka[s-1+(i-1)*s]+l1;
	setka[s*(s-1)+i]+=setka[s*(s-1)+i-1]+l1;
	arr[i]=setka[i];
	arr[i*s]=setka[i*s];
	arr[s-1+i*s]=setka[s-1+i*s];
	arr[s*(s-1)+i]=setka[s*(s-1)+i];
//	arr2[i]=setka[i];
//	arr2[i*s]=setka[i*s];
//	arr2[s-1+i*s]=setka[s-1+i*s];
//	arr2[s*(s-1)+i]=setka[s*(s-1)+i];
    }

    if(s<16)
    {
      for(int i=0; i<s; i++)
      {
        for(int j=0; j<s; j++)
          printf("%f ",setka[IDX2C(i,j,s)]);
        printf("\n");
      
      }
    }

    hipblasStatus_t status;
    hipblasHandle_t handle;
    printf("%d\n",HIPBLAS_STATUS_MAPPING_ERROR);

    #pragma acc data copyin(setka[0:s*s],arr[0:s*s],arr2[0:s*s],setka2[0:s*s],err,iter)
	{
    status = hipblasCreate(&handle);
	  if (status != HIPBLAS_STATUS_SUCCESS)
	  	printf("ERROR: %d!\n",status);
    while(err>a && iter<n)
    {
      iter++;
	if(iter%100==0 || iter==1)
	{
      #pragma acc kernels
      {
        err=0;
      }
	}
#pragma acc data present(arr, setka)
#pragma acc parallel loop gang num_workers(4) vector_length(128) async(1)
      for(int i=1; i<s-1; i++)
      {
        #pragma acc loop vector 
        for(int j=1; j<s-1; j++)
        {
          arr[IDX2C(i,j,s)]=0.25*(setka[IDX2C(i,j-1,s)]+setka[IDX2C(i,j+1,s)]+setka[IDX2C(i-1,j,s)]+setka[IDX2C(i+1,j,s)]);
        }
      }
      float* dop;
      dop = arr;
      arr=setka;
      setka = dop;
	  #pragma acc update device(setka,arr) async(1)
      if(iter%100==0 || iter==1)
      {
        int nm=0;
        float alpha=-1;
        #pragma acc host_data use_device(arr,setka,arr2)
        {
    //	  #pragma acc update host(setka,arr) async(1)
  //      status=hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
          status=hipblasScopy(handle,s*s,arr,1,arr2,1);
          if (status != HIPBLAS_STATUS_SUCCESS)
            printf("ERROR 1: %d!\n",status);
#pragma acc kernels
	  {
	  if(s<16)
	      for(int i=0; i<s; i++)
	      {
		for(int j=0; j<s; j++)
		  printf("%f ",arr2[IDX2C(i,j,s)]);
		printf("\n");
	      }
	  }
          status=hipblasSaxpy(handle,s*s,&alpha,setka,1,arr2,1);
          if (status != HIPBLAS_STATUS_SUCCESS)
            printf("ERROR 2: %d!\n",status);
          status=hipblasIsamin(handle,s*s,arr2,1,&nm);
          if (status != HIPBLAS_STATUS_SUCCESS)
            printf("ERROR 3: %d!\n",status);
          #pragma acc kernels
          {
          err = fabs(arr[nm-1]);
          }
          #pragma acc update host(err) async(1)
          #pragma acc wait(1)
          printf("%d %f\n",iter, err);
        }
        
//        #pragma acc wait(1) 
      }
    }
    status=hipblasDestroy(handle);
    }
    printf("Count iterations: %d\nError: %.10f\n", iter,err);
	  if (status != HIPBLAS_STATUS_SUCCESS)
	  	printf("ERROR: %d!\n",status);
    if(s<16)
      for(int i=0; i<s; i++)
      {
        for(int j=0; j<s; j++)
          printf("%f ",setka[IDX2C(i,j,s)]);
        printf("\n");
      }
    free(arr);
    free(setka);
    free(setka2);
    free(arr2);
  }
  return 0;
}
