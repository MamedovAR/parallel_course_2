#include <hipcub/hipcub.hpp>
#include <iostream>
#include <stdlib.h>

int main(){


  // Declare, allocate, and initialize device-accessible pointers for input and output
  int                      num_items = 32;
  double                      *d_in;
  double   *d_out;

  double *h_in = (double*)malloc(32*sizeof(double));//new double[num_items];
  double *h_out = (double*)malloc(sizeof(double));
  hipMalloc(&d_in, num_items*sizeof(d_in[0]));
  hipMalloc(&d_out, sizeof(double));
  for (int i = 0; i < num_items; i++) h_in[i] = (double)i;
  h_in[12] = 8;  // so we expect our return tuple to be 12,2
  hipMemcpy(d_in, h_in, num_items*sizeof(d_in[0]), hipMemcpyHostToDevice);

  // Determine temporary device storage requirements
  void     *d_temp_storage = NULL;
  size_t   temp_storage_bytes = 0;
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run argmin-reduction
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);

  hipMemcpy(h_out, d_out, sizeof(double), hipMemcpyDeviceToHost);
  printf("%f\n",h_out[0]);
//  std::cout << "maximum value: " << h_out[0] << std::endl;
  return 0;
}