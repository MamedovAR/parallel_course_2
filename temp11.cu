#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipcub/hipcub.hpp>
#include <nvToolsExt.h>
#include <hip/hip_runtime.h>
#include "/opt/nvidia/hpc_sdk/Linux_x86_64/22.11/math_libs/11.0/targets/x86_64-linux/include/hipblas.h"
#define IDX2F(i,j,ld) (((j)-1)*(ld))+((i)-1)
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

__device__ void change(float* setka, float* arr, int i, int j, int s)
{
	setka[IDX2C(i+threadIdx.x,j+threadIdx.y,s)]=0.25*(arr[IDX2C(i+threadIdx.x,j-1+threadIdx.y,s)]+arr[IDX2C(i+threadIdx.x,j+1+threadIdx.y,s)]+arr[IDX2C(i-1+threadIdx.x,j+threadIdx.y,s)]+arr[IDX2C(i+1+threadIdx.x,j+threadIdx.y,s)]);
}

/*_device__ void init(int s, int i, float l1, float l2)
{
	setka[i]=setka[i-1]+l1;
	setka[i*s]+=setka[(i-1)*s]+l2;
	setka[s-1+i*s]+=setka[s-1+(i-1)*s]+l1;
	setka[s*(s-1)+i]+=setka[s*(s-1)+i]+l1;
}*/

__device__ void deliter(float* setka, float* arr, float* err, int i)
{
	err[i+threadIdx.x]=setka[i+threadIdx.x]-arr[i+threadIdx.x];
}

int main(int argc, char** argv)
{
	float a=0;
	int s=0;
	int n=0;
	hipblasStatus_t status;
	hipblasHandle_t handle;
	if(argv[1][1]=='h')
	{
		printf("Put -h to show this.\n");
		printf("Put -a <NUMBER_OF_ACCURACY*10^6> -s <SIZE^2> -n <NUMBER_OF_ITERATION*10^6>.\n");
	}
	else
	{
		for(int k=1; k<argc; k+=2)
		{
			if(argv[k][1]=='a')
				a=(float)atof(argv[k+1]);
			else if(argv[k][1]=='s')
				s=atoi(argv[k+1]);
			else if(argv[k][1]=='n')
				n=atoi(argv[k+1]);
		}
		float* setka;
		hipMalloc((void**)setka,s*s*sizeof(float));
		float* arr;
		hipMalloc((void**)arr,s*s*sizeof(float));
		float* errors;
		hipMalloc((void**)errors,s*s*sizeof(float));
		setka[0]=10;
		setka[s-1]=20;
		setka[(s-1)*s]=30;
		setka[s*s-1]=20;
		float l1=(10);
		l1/=s;
		float l2=20;
		l2/=s;
		for(int i=1; i<s-1; i+=32)
		{
//			init<<<2,5>>>(s,i,l1,l2);
			setka[i]=setka[i-1]+l1;
			setka[i*s]+=setka[(i-1)*s]+l2;
			setka[s-1+i*s]+=setka[s-1+(i-1)*s]+l1;
			setka[s*(s-1)+i]+=setka[s*(s-1)+i]+l1;
		}
		hipDeviceSynchronize();
		int iter=0;
		float err=1;
//		cublasInit();
		while(err>a && iter<n)
		{
			iter++;
			err=0;
			for(int i=0; i<s*s; i++)
			{
				arr[i]=setka[i];
			}
			for(int i=1; i<s-1; i+=32)
			{
				for(int j=1; j<s-1; j+=32)
				{
//					setka[i+j*(s-1)]=0.25*(arr[i+1+j*(s-1)]+arr[i-1+j*(s-1)]+arr[i+(j-1)*(s-1)]+arr[i+(j+1)*(s-1)]);
					change<<<2,32>>>(setka,arr,i,j,s);
				}
			}
			for(int i=0; i<s*s; i+=32
				deliter<<<2,32>>>(setka,arr,errors,i);
			for(int i=0; i<s*s; i++)
				if(err<errors[i])
					err=errors[i];
			//
			if(iter%100==0 || iter==1)
				printf("%d %f \n",iter, err);
		}
		hipFree(arr);
		hipFree(setka);
		hipFree(errors);
		hipblasDestroy(handle);
//		cublasShutdown();
		printf("Count iterations: %d", iter);
		free(setka);
	}
	return 0;
}
