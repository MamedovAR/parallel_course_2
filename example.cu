#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main()
{
    const int n = 3, m = 4; // размер матрицы
    const float alpha = 1.0f, beta = 0.0f;
    float *h_A, *h_x, *h_y;
    float *d_A, *d_x, *d_y;
    hipblasHandle_t handle;

    // выделение памяти на хосте
    h_A = (float*)malloc(n * n * sizeof(float));
    h_x = (float*)malloc(n * sizeof(float));
    h_y = (float*)malloc(n * sizeof(float));

    // заполнение матрицы и векторов
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            h_A[i * n + j] = 0.5;//i + j;
        }
        h_x[i] = 0.5;//i;
//        h_y[i] = 1;//0;
    }

    // выделение памяти на устройстве
    hipMalloc((void**)&d_A, n * m * sizeof(float));
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, m * sizeof(float));

    // копирование данных на устройство
    hipMemcpy(d_A, h_A, n * m * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, m * sizeof(float), hipMemcpyHostToDevice);

    // создание объекта handle для работы с cuBLAS
    hipblasCreate(&handle);

    // выполнение умножения матрицы на вектор
    hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha, d_A, m, d_x, 1, &beta, d_y, 1);

    // копирование результата на хост
    hipMemcpy(h_y, d_y, m * sizeof(float), hipMemcpyDeviceToHost);

    // вывод результата
    for (int i = 0; i < n; i++) {
        printf("%f ", h_y[i]);
    }

    // освобождение памяти
    free(h_A);
    free(h_x);
    free(h_y);
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    hipblasDestroy(handle);

    return 0;
}